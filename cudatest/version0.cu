//version0


#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 2048 // WIDTH x WIDTH matrix
#define TILE_WIDTH 32 //tile width, should be TILE_WIDTH * TILE_WIDTH < 1024 (maxmum thread for 0ne block)

void MatrixMultiplication (float *M, float *N, float *P, int Width) {
        for (int i = 0; i < Width; ++i)
                for (int j= 0; j < Width; ++j) {
                        float sum = 0;
                        for (int k = 0; k < Width; ++k) {
                                float a = M[i * Width + k] ;
                                float b = N[k * Width + j] ;
                                sum += a * b ;
                        }
                        P[i * Width + j] = sum ;
                }
}

int main () {
        float *M, *N, *P ;
        int k ;
        int size = WIDTH*WIDTH*sizeof(float) ;

        hipEvent_t start, stop ;
        float time, cpu_time;

        hipEventCreate (&start) ;
        hipEventCreate (&stop) ;
        M = (float *) malloc (size) ;
        N = (float *) malloc (size) ;
        P = (float *) malloc (size) ;

        for (k=0; k<WIDTH; ++k) {
                M[k] = 1. ;
                N[k*WIDTH] = 1. ;
        }

        hipEventRecord (start,0) ;
        MatrixMultiplication (M, N, P, WIDTH) ;

        hipEventRecord (stop, 0) ;
        hipEventSynchronize(stop) ;
        hipEventElapsedTime (&time, start, stop) ;
        cpu_time = time ;
        printf ("CPU time=%f msec\n",cpu_time) ;
        printf ("Fig 3.4: P[0]=%f\n", P[0]) ; P[0] = -1 ;

        hipMalloc ((void**) &M, size) ;
        hipMalloc ((void**) &N, size) ;
        hipMalloc ((void**) &P, size) ;

        hipFree (M) ;
        hipFree (N) ;
        hipFree (P) ;


}
